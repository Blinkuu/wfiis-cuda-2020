#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include "timer.h"

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively
 * and use profiler to check your progress
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 25us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * __restrict__ a, int * __restrict__ b, int * __restrict__ result)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

//	if(index < N)
//		result[index] = 2 * a[index] + b[index];

	const unsigned int s = blockDim.x * gridDim.x;
		while( i + s * 2 < N )
		{
			result[i] = 2 * a[i] + b[i];
			i += s;
			result[i] = 2 * a[i] + b[i];
			i += s;
			result[i] = 2 * a[i] + b[i];
			i += s;
		}
		while(	i < N  	)
		{
			result[i] = 2 * a[i] + b[i];
			i += s;
		}

}


__global__ void init(int * a, int val)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if ( tid < N )
        a[tid] = val;
}


int main()
{
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    int deviceId;
	hipGetDevice(&deviceId);

    cuda::timer::start();

//    a = (int*) malloc(size);
//    b = (int*) malloc(size);
//    c = (int*) malloc(size);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

	hipMemPrefetchAsync(a, size, deviceId);
	hipMemPrefetchAsync(b, size, deviceId);
	hipMemPrefetchAsync(c, size, deviceId);

    int threads_per_block = 64;
    int number_of_blocks = 256; //(N + threads_per_block - 1) / threads_per_block;//(N / threads_per_block) + 1;

    init <<< number_of_blocks, threads_per_block >>> ( a, 2 );
	init <<< number_of_blocks, threads_per_block >>> ( b, 1 );
	init <<< number_of_blocks, threads_per_block >>> ( c, 0 );

	saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );


    hipDeviceSynchronize();
//    for(int i = 0; i < N; i++){
//    	a[i] = 2;
//    	b[i] = 1;
//    	c[i] = a[i] * 2 + b[i];
//    }

    cuda::timer::stop();
    printf("time: %f s\n", cuda::timer::read());

    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    //hipFree( a ); hipFree( b ); hipFree( c );

}

