#include "hip/hip_runtime.h"
/// managed mamory analysis - cuda lab gpu only mamory access

#include "page_faults.h"
#include <iostream>

__global__
static void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

static void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

void test_page_fault_gpu_only()
{
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);
  cuda::timer::start();
  deviceKernel<<<256, 256>>>(a, N);
  hipDeviceSynchronize();

  hipFree(a);
  cuda::timer::stop();
  std::cout << "[DONE] test_page_fault_gpu_only\n";
  std::cout << "[TIME] " << cuda::timer::read() << " s\n";
}

